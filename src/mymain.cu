#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include "hip/hip_runtime.h"
#include "structs.h"
#include "device_scan.cuh"
#include "find_score.cuh"

char *get_protein(char *filename, int *s, char reset) {
	FILE *f = fopen(filename, "r");
	char c;
	char *protein;
	int ignore;
	int it, size;
	
	rewind(f);
	size = 0;
	while(fscanf(f, "%c", &c) != EOF) {
		printf("size = %d, reading c = '%c'\n", size, c);
		if (c == '>') {
			ignore = 1;
			if (size > 0)
				++size;
		}
		else if (ignore && c == '\n')
			ignore = 0;
		else if (!ignore && c >= 'A' && c <= 'Z')
			++size;
	}
	*s += size;
	printf("total size: %d\n\n", *s);
	fflush(stdout);
	
	protein = (char *)malloc((size + 1) * sizeof(char));
	it = 0;
	rewind(f);
	while(fscanf(f, "%c", &c) != EOF) {
		if (c == '>') {
			ignore = 1;
			if (it > 0) {
				*(protein + it) = reset;
				++it;
			}
		}
		else if (ignore && c == '\n')
			ignore = 0;
		else if (!ignore && c >= 'A' && c <= 'Z') {
			if (it >= size) {
				printf("char * size not enough!\n");
				fflush(stdout);
				return NULL;
			}
			*(protein + it) = c;
			++it;
		}
	}
	*(protein + size) = '\0';
	fclose(f);
	
	return protein;
}

void init(data *mat, int val, int size) {
	data->N = (int *)malloc(size * sizeof(int));
	data->H = (int *)malloc(size * sizeof(int));
	data->V = (int *)malloc(size * sizeof(int));
	
	for (int i = 0; i < size; ++i) {
		*(mat->N + i) = 0;
		*(mat->H + i) = 0;
		*(mat->V + i) = 0;
	}
}

int main(int argc, char **argv) {
	//variableinit
	
	printf("Welcome!\n");
	printf("This is a protein alignment software. The expected input is \n");
	printf("a list of files that contain protein representations.\n");
	
	if (argc < 3) {
		printf("I expect to receive at least two filenames as arguments.\n");
		exit(-1);
	}
	
	
//	printf("reset character?\n> ");
//	scanf(" %c", &(config.reset));
	
	int v_len = 0;	
	char *vertical = get_protein(argv[1], &v_len, config.reset);
	
	int h_len = 0;
	char *horizontal = get_protein(argv[2], &h_len, config.reset);
	for (int i = 3; i < argc; ++i) {
		strcat(horizontal, &(config.reset));
		++h_len;
		strcat(horizontal, get_protein(argv[i], &h_len, config.reset));
	}
	
	int mat_len = h_len + 1;
	configuration config;
	config.reset = '#';
	config.thread_chunk = 256;
	config.block_size = min(512, (mat_len + config.thread_chunk - 1) / 
														config.thread_chunk);
	config.grid_size = (mat_len + config.block_size * config.thread_chunk - 1) / 
									(config.block_size * config.thread_chunk);
	
	data matRow[2];//TODO: set up
	data devMatRow[2];//TODO: set up
	
	int *auxiliary = (int *)malloc(config.grid_size);
	int devAux;//set up
	
	return 0;
}
